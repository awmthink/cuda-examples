#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>

#include "common/cuda_helper.h"
#include "common/random.h"
#include "common/stopwatch.h"

constexpr int kBlockSize = 1024;

// 1, 3, 6, 10, 15   6, 13, 21, 30
template <typename T>
void BlellochScanCPU(T *in, T *out, int n) {
  int s = 0;
  for (s = 2; s <= n; s = s * 2) {
    for (int i = s - 1; i < n; i += s) {
      in[i] += in[i - s / 2];
    }
  }
  in[s / 2 - 1] = 0;
  for (s = n; s > 1; s = s / 2) {
    for (int i = s - 1; i < n; i += s) {
      float temp = in[i] + in[i - s / 2];
      in[i - s / 2] = in[i];
      in[i] = temp;
    }
  }
  for (int i = 0; i < n; ++i) {
    out[i] = in[i];
  }
}

template <typename T>
__global__ void blelloch_scan_kernel(const T *data, T *out) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  int tid = threadIdx.x;

  extern __shared__ T sdata[];
  sdata[tid] = data[gid];

  for (int stride = 2; stride <= blockDim.x; stride *= 2) {
    __syncthreads();
    int idx = (tid + 1) * stride - 1;
    if (idx < blockDim.x) {
      sdata[idx] += sdata[idx - stride / 2];
    }
  }
  sdata[blockDim.x - 1] = 0;

  for (int stride = blockDim.x; stride >= 2; stride /= 2) {
    __syncthreads();
    int idx = (tid + 1) * stride - 1;
    if (idx < blockDim.x) {
      T temp1 = sdata[idx] + sdata[idx - stride / 2];
      T temp2 = sdata[idx];
      __syncthreads();
      sdata[idx] = temp1;
      sdata[idx - stride / 2] = temp2;
    }
  }
  out[gid] = sdata[tid];
}

// 代码只显示了一个block中的scan
template <typename T>
void BlellochScanGPU(T *vec, T *out, int n) {
  T *d_vec = nullptr;
  T *d_out = nullptr;

  int blocks = (n + kBlockSize - 1) / kBlockSize;
  int n_bytes = sizeof(T) * n;
  // 按kBlockSize对齐，进行超分配，末部全部填充0，不影响scan结果
  checkCudaErrors(hipMalloc(&d_vec, sizeof(T) * blocks * kBlockSize));
  checkCudaErrors(hipMemset(d_vec, 0, sizeof(T) * blocks * kBlockSize));
  checkCudaErrors(hipMemcpy(d_vec, vec, n_bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&d_out, sizeof(T) * blocks * kBlockSize));

  blelloch_scan_kernel<<<blocks, kBlockSize, kBlockSize * sizeof(T)>>>(d_vec, d_out);

  checkCudaErrors(hipMemcpy(out, d_out, n_bytes, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_vec));
  checkCudaErrors(hipFree(d_out));
}

constexpr bool IsPowerofTwo(int n) {
  if (n < 0) {
    return false;
  }
  return (n & (n - 1)) == 0;
}

int main() {
  constexpr int kNumSize = 32;
  static_assert(IsPowerofTwo(kNumSize), "kNumSize must be power of 2");
  std::vector<int> vec(kNumSize);
  FillSequenceNumbers(vec, 1);
  std::cout << "Origin Vector: ";
  PrintElements(vec);

  std::vector<int> out(kNumSize);
  BlellochScanCPU(vec.data(), out.data(), kNumSize);
  std::cout << "Scan Vecotr: ";
  PrintElements(out);

  // reset data
  FillSequenceNumbers(vec, 1);
  std::fill(out.begin(), out.end(), 0);

  static_assert(kNumSize < kBlockSize * kBlockSize, "NumSize is too large");
  BlellochScanGPU(vec.data(), out.data(), kNumSize);
  std::cout << "GPU Scan Vecotr: ";
  PrintElements(out);

  return 0;
}
