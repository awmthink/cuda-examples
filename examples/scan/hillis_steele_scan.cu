#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>

#include "common/cuda_helper.h"
#include "common/random.h"
#include "common/stopwatch.h"

constexpr int kBlockSize = 1024;

template <typename T>
void HillisSteeleScanCPU(T *in, T *out, int n) {
  T *output = out;
  for (int s = 1; s < n; s = s * 2) {
    for (int i = 0; i < n; i++) {
      if (i - s >= 0) {
        out[i] = in[i] + in[i - s];
      } else {
        out[i] = in[i];
      }
    }
    std::swap(in, out);
  }
  // 经过迭代后，最终scan的结果是存储在in数组中的，将其拷贝到原始的输入数组中
  for (int i = 0; i < n; ++i) {
    output[i] = in[i];
  }
}

template <typename T>
__global__ void hillis_steele_scan_kernel(T *in, T *out, T *seg_out) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  // 申请一片 2*blockDim.x 大小的共享内存，然后切成2份
  extern __shared__ T sdata[];
  T *sdata_in = &sdata[0];
  T *sdata_out = &sdata[blockDim.x];
  // sout在每一step后存储最终的输出
  T *sout = sdata_out;

  sdata_in[tid] = in[gid];
  __syncthreads();

  for (int s = 1; s < blockDim.x; s *= 2) {
    if (tid - s >= 0) {
      sdata_out[tid] = sdata_in[tid] + sdata_in[tid - s];
    } else {
      sdata_out[tid] = sdata_in[tid];
    }
    __syncthreads();
    sout = sdata_out;
    sdata_out = sdata_in;
    sdata_in = sout;
  }
  out[gid] = sout[tid];

  // 将每个block中最后一个结果，写入到seg_out中
  if (seg_out != nullptr && tid == blockDim.x - 1) {
    seg_out[blockIdx.x] = sout[tid];
  }
}

template <typename T>
__global__ void scan_add_segment_kernel(T *data, T *seg_data) {
  if (blockIdx.x > 0) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    data[gid] += seg_data[blockIdx.x - 1];
  }
}

template <typename T>
void HillisSteeleScanGPU(T *vec, T *out, int n) {
  T *d_vec = nullptr;
  T *d_out = nullptr;
  T *d_seg_out = nullptr;  // 存储每一个block的scan结果的最后一个元素

  int blocks = (n + kBlockSize - 1) / kBlockSize;
  int sub_blocks = (blocks + kBlockSize - 1) / kBlockSize;

  int n_bytes = sizeof(T) * n;
  // 按kBlockSize对齐，进行超分配，末部全部填充0，不影响scan结果
  checkCudaErrors(hipMalloc(&d_vec, sizeof(T) * blocks * kBlockSize));
  checkCudaErrors(hipMemset(d_vec, 0, sizeof(T) * blocks * kBlockSize));
  checkCudaErrors(hipMemcpy(d_vec, vec, n_bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc(&d_out, n_bytes));

  checkCudaErrors(hipMalloc(&d_seg_out, sub_blocks * kBlockSize * sizeof(T)));

  hillis_steele_scan_kernel<<<blocks, kBlockSize, 2 * kBlockSize * sizeof(T)>>>(d_vec, d_out,
                                                                                d_seg_out);
  hillis_steele_scan_kernel<<<sub_blocks, kBlockSize, 2 * kBlockSize * sizeof(T)>>>(
      d_seg_out, d_vec, static_cast<T *>(nullptr));

  scan_add_segment_kernel<<<blocks, kBlockSize>>>(d_out, d_vec);

  checkCudaErrors(hipMemcpy(out, d_out, n_bytes, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_vec));
  checkCudaErrors(hipFree(d_out));
  checkCudaErrors(hipFree(d_seg_out));
}

int main() {
  constexpr int kNumSize = 10;
  std::vector<int> vec(kNumSize);
  FillSequenceNumbers(vec, 1);
  std::cout << "Origin Vector: ";
  PrintElements(vec);

  std::vector<int> out(kNumSize);
  HillisSteeleScanCPU(vec.data(), out.data(), kNumSize);
  std::cout << "Scan Vecotr: ";
  PrintElements(out);

  // reset data
  FillSequenceNumbers(vec, 1);
  std::fill(out.begin(), out.end(), 0);

  static_assert(kNumSize < kBlockSize * kBlockSize, "NumSize is too large");
  HillisSteeleScanGPU(vec.data(), out.data(), kNumSize);
  std::cout << "GPU Scan Vecotr: ";
  PrintElements(out);

  return 0;
}