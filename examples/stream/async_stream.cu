#include "hip/hip_runtime.h"
#include <iostream>
#include "common/cuda_helper.h"
#include "common/stopwatch.h"

constexpr int kElemNum = 1'000'000;

__global__ void simple_square_kernel(float *data, int n) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < n) {
    float temp = data[gid];
    data[gid] = temp * temp;
  }
}

__global__ void simple_exp_kernel(float *data, int n) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < n) {
    float temp = data[gid];
    data[gid] = __expf(temp);
  }
}

void CUDASyncStream() {
  hipStream_t stream;
  hipStreamCreate(&stream);

  std::size_t sz = kElemNum * sizeof(float);

  float *h_a, *h_b;
  checkCudaErrors(hipHostAlloc(&h_a, sz, hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc(&h_b, sz, hipHostMallocDefault));

  float *d_a, *d_b;
  checkCudaErrors(hipMalloc(&d_a, sz));
  checkCudaErrors(hipMalloc(&d_b, sz));

  int threads = 512;
  int blocks = (kElemNum + threads - 1) / threads;

  Stopwatch sync_timer;
  sync_timer.Start();

  checkCudaErrors(hipMemcpyAsync(d_a, h_a, sz, hipMemcpyHostToDevice, stream));
  simple_square_kernel<<<blocks, threads, 0, stream>>>(d_a, kElemNum);
  checkCudaErrors(hipMemcpyAsync(h_a, d_a, sz, hipMemcpyDeviceToHost, stream));

  checkCudaErrors(hipMemcpyAsync(d_b, h_b, sz, hipMemcpyHostToDevice, stream));
  simple_exp_kernel<<<blocks, threads, 0, stream>>>(d_b, kElemNum);
  checkCudaErrors(hipMemcpyAsync(h_b, d_b, sz, hipMemcpyDeviceToHost, stream));

  checkCudaErrors(hipStreamSynchronize(stream));

  std::cout << "sync time: " << sync_timer.Elapsed<Stopwatch::MICROSECONDS>() << " us\n";

  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_b));
  checkCudaErrors(hipStreamDestroy(stream));
}

void CUDAAsyncStream() {
  hipStream_t stream1, stream2;
  checkCudaErrors(hipStreamCreate(&stream1));
  checkCudaErrors(hipStreamCreate(&stream2));

  std::size_t sz = kElemNum * sizeof(float);

  float *h_a, *h_b;
  checkCudaErrors(hipHostAlloc(&h_a, sz, hipHostMallocDefault));
  checkCudaErrors(hipHostAlloc(&h_b, sz, hipHostMallocDefault));

  float *d_a, *d_b;
  checkCudaErrors(hipMalloc(&d_a, sz));
  checkCudaErrors(hipMalloc(&d_b, sz));

  int threads = 512;
  int blocks = (kElemNum + threads - 1) / threads;

  Stopwatch sync_timer;
  sync_timer.Start();

  checkCudaErrors(hipMemcpyAsync(d_a, h_a, sz, hipMemcpyHostToDevice, stream1));
  simple_square_kernel<<<blocks, threads, 0, stream1>>>(d_a, kElemNum);
  checkCudaErrors(hipMemcpyAsync(h_a, d_a, sz, hipMemcpyDeviceToHost, stream1));

  checkCudaErrors(hipMemcpyAsync(d_b, h_b, sz, hipMemcpyHostToDevice, stream2));
  simple_exp_kernel<<<blocks, threads, 0, stream2>>>(d_b, kElemNum);
  checkCudaErrors(hipMemcpyAsync(h_b, d_b, sz, hipMemcpyDeviceToHost, stream2));

  checkCudaErrors(hipStreamSynchronize(stream1));
  checkCudaErrors(hipStreamSynchronize(stream2));

  std::cout << "async time: " << sync_timer.Elapsed<Stopwatch::MICROSECONDS>() << " us\n";

  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_b));
  checkCudaErrors(hipStreamDestroy(stream1));
  checkCudaErrors(hipStreamDestroy(stream2));
}

int main() {
  // copy A -> compute A -> copy back A -> copy B -> compute B -> copy back B
  CUDASyncStream();

  // copy A -> compute A -> copy back A
  //   copy B -> compute B -> copy back B
  CUDAAsyncStream();

  return 0;
}