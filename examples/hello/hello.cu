
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_runtime_api.h>

__global__ void hello() { printf("hello, "); }

int main(int argc, char *argv[]) {
  hello<<<1, 1>>>();
  hipDeviceSynchronize();
  printf("world!\n");
}